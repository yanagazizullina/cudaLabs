#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <time.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>
using namespace std;

#define N 1000
#define blocks 32
#define threads 32

__global__ void gpu_calculation_pi(float* estimate, hiprandState* states) {
	unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
	int points_in_circle = 0;
	float x, y;
	hiprand_init(1234, tid, 0, &states[tid]);  
	for (int i = 0; i < N; i++) {
		x = hiprand_uniform(&states[tid]);
		y = hiprand_uniform(&states[tid]);
		points_in_circle += (x * x + y * y <= 1.0f); 
	}
	estimate[tid] = 4.0f * points_in_circle / (float)N;
}

float cpu_calculation_pi(long trials) {
	float x, y;
	long points_in_circle = 0.0f;
	for (long i = 0; i < trials; i++) {
		x = rand() / (float)RAND_MAX;
		y = rand() / (float)RAND_MAX;
		points_in_circle += (x * x + y * y <= 1.0f);
	}
	return 4.0f * points_in_circle / trials;
}

int main(int argc, char* argv[]) {
	clock_t cpu_start, cpu_stop;
	float host[blocks * threads];
	float* device;
	hiprandState* deviceStates;
	float gpu_time;
	hipEvent_t gpu_start, gpu_stop;

	hipEventCreate(&gpu_start);
	hipEventCreate(&gpu_stop);
	hipMalloc((void**)&device, blocks * threads * sizeof(float)); 
	hipMalloc((void**)&deviceStates, threads * blocks * sizeof(hiprandState));
	hipEventRecord(gpu_start, 0);
	gpu_calculation_pi << <blocks, threads >> > (device, deviceStates);
	hipEventRecord(gpu_stop, 0);
	hipEventSynchronize(gpu_stop);
	hipEventElapsedTime(&gpu_time, gpu_start, gpu_stop);
	hipMemcpy(host, device, blocks * threads * sizeof(float), hipMemcpyDeviceToHost); 
	
	float gpu_pi = 0.0f;
	for (int i = 0; i < blocks * threads; i++) {
		gpu_pi += host[i];
	}

	gpu_pi /= (blocks * threads);

	cout << "Approximate pi calculated on GPU is: " << gpu_pi << " and calculation took " << gpu_time << " msec" << endl;

	hipFree(device);
	hipFree(deviceStates);

	cpu_start = clock();
	float cpu_pi = cpu_calculation_pi(blocks * threads * N);
	cpu_stop = clock();
	cout << "Approximate pi calculated on CPU is: " << cpu_pi << " and calculation took " << (cpu_stop - cpu_start) / double(CLOCKS_PER_SEC) * 1000 << " msec" << endl;

	return 0;
}